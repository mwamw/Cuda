#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>

using namespace std;

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
#define BLOCK_SIZE 8

// CUDA 错误检查宏
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at line %d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while (0)

// GPU 核心计算函数
__global__ void life3d_kernel(const char* universe, char* next, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (idx >= N || idy >= N || idz >= N) return;
    
    int alive = 0;
    // 计算邻居存活数量
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0) continue;
                
                int nx = (idx + dx + N) % N;
                int ny = (idy + dy + N) % N;
                int nz = (idz + dz + N) % N;
                
                alive += universe[nx * N * N + ny * N + nz];
            }
        }
    }
    
    // 应用生命游戏规则
    int current = universe[idx * N * N + idy * N + idz];
    if (current && (alive < 5 || alive > 7))
        next[idx * N * N + idy * N + idz] = 0;
    else if (!current && alive == 6)
        next[idx * N * N + idy * N + idz] = 1;
    else
        next[idx * N * N + idy * N + idz] = current;
}

// 存活细胞数（在 CPU 上计算）
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// GPU 版本的生命游戏主函数
void life3d_run(int N, char *universe, int T)
{
    // 分配 GPU 内存
    char *d_universe, *d_next;
    size_t size = N * N * N * sizeof(char);
    
    CHECK_CUDA_ERROR(hipMalloc(&d_universe, size));
    CHECK_CUDA_ERROR(hipMalloc(&d_next, size));
    CHECK_CUDA_ERROR(hipMemcpy(d_universe, universe, size, hipMemcpyHostToDevice));
    
    // 设置 CUDA 网格和块的大小
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + block.x - 1) / block.x,
              (N + block.y - 1) / block.y,
              (N + block.z - 1) / block.z);
    
    // 主循环
    for (int t = 0; t < T; t++) {
        life3d_kernel<<<grid, block>>>(d_universe, d_next, N);
        CHECK_CUDA_ERROR(hipGetLastError());
        
        // 交换指针
        char* temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }
    
    // 将结果复制回 CPU
    CHECK_CUDA_ERROR(hipMemcpy(universe, d_universe, size, hipMemcpyDeviceToHost));
    
    // 释放 GPU 内存
    CHECK_CUDA_ERROR(hipFree(d_universe));
    CHECK_CUDA_ERROR(hipFree(d_next));
}

// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size)) {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file) {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    if (argc < 5) {
        cout << "usage: ./life3d_gpu N T input output" << endl;
        return 1;
    }
    
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];
    
    char *universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);
    
    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    
    life3d_run(N, universe, T);
    
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    
    write_file(output_file, universe, N);
    
    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;
    
    free(universe);
    return 0;
}